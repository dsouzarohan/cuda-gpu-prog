# include <hip/hip_runtime.h>
# include <stdio.h>

__global__ void vector_add(int* a, int* b, int* c, int N) {
    int i = threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    // printf("Vector addition on GPU");
    int h_arr_A[] = {1,2,3,4,5};
    int h_arr_B[] = {2,4,5,6,10};
    int* h_arr_C;

    size_t size = sizeof(h_arr_A) / sizeof(h_arr_A[0]);
    printf("Size of arrays is %zu\n", size);

    int* d_arr_A;
    int* d_arr_B;
    int* d_arr_C; // device pointer

    // printf("cudaMalloc d_arr_A");
    hipMalloc((void**) &d_arr_A, size * sizeof(int));
    // printf("cudaMalloc d_arr_B");
    hipMalloc((void**) &d_arr_B, size * sizeof(int));
    // printf("cudaMalloc d_arr_C");
    hipMalloc((void**) &d_arr_C, size * sizeof(int));

    hipMemcpy(d_arr_A, h_arr_A, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr_B, h_arr_B, size * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 grid_size(1);
    dim3 block_size(size);

    // printf("Launching kernel vector_add");
    vector_add<<<grid_size, block_size>>>(d_arr_A, d_arr_B, d_arr_C, size);

    hipDeviceSynchronize();

    h_arr_C = (int*) malloc(size * sizeof(int));

    // printf("Copying back to host");
    hipMemcpy(h_arr_C, d_arr_C, size * sizeof(int), hipMemcpyDeviceToHost);

    // printf("Freeing from GPU\n");
    hipFree(d_arr_A); hipFree(d_arr_B); hipFree(d_arr_C);

    for(int i = 0; i < size; i++) {
        printf("Value at %p is %d\n", h_arr_C, h_arr_C[i]);
    }
    return 0;
}